#include "hip/hip_runtime.h"
#include <time.h>
#include <hip/hip_runtime.h>
#include <>
#include <opencv2/cudaarithm.hpp>
#include <opencv2/cudafilters.hpp>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include "opencv2/cudaimgproc.hpp"
#include "matchTracker.h"
#include "Particle.h"

__global__ void genericErodeKernel(cv::cuda::GpuMat out, cv::cuda::GpuMat flameFrame, cv::cuda::GpuMat fullFrame, int *alphas)
{
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	int x = WINDOW_WIDTH;
	int y = WINDOW_HEIGHT;
	if (threadId < x * y)
	{
		int row = threadId / x;
		int column = threadId % x;
		uint8_t pixelR = flameFrame.data[(row*flameFrame.step) + column * 3 + 2];
		uint8_t pixelG = flameFrame.data[(row*flameFrame.step) + column * 3 + 1];
		uint8_t pixelB = flameFrame.data[(row*flameFrame.step) + column * 3];

		bool allPixelsFire = false;
		if ((pixelR + pixelG + pixelB) > 0)
		{
			allPixelsFire = true;
			for (int i = -2; i < 3; i++)
			{
				for (int j = -2; j < 3; j++)
				{
					if ((row + i > -1) && (row + i < y) && (column + j > -1) && (column + j < x))
					{
						if ((flameFrame.data[((row + i)*flameFrame.step) + (column + j) * 3 + 2] == 0)&& (flameFrame.data[((row + i)*flameFrame.step) + (column + j) * 3 + 1] == 0)&& (flameFrame.data[((row + i)*flameFrame.step) + (column + j) * 3] == 0))
						{
							allPixelsFire = false;
						}
					}
				}
			}
		}
		if (allPixelsFire)
		{
			out.data[(row*out.step) + column * 3] = pixelB * float(alphas[row * x + column])/255 + (1 - float(alphas[row * x + column])/255) * fullFrame.data[(row*fullFrame.step) + column * 3];
			out.data[(row*out.step) + column * 3 + 1] = pixelG * float(alphas[row * x + column])/255 + (1 - float(alphas[row * x + column])/255) * fullFrame.data[(row*fullFrame.step) + column * 3 + 1];
			out.data[(row*out.step) + column * 3 + 2] = pixelR * float(alphas[row * x + column])/255 + (1 - float(alphas[row * x + column])/255) * fullFrame.data[(row*fullFrame.step) + column * 3 + 2];
		}
		else {
			out.data[(row*out.step) + column * 3 + 2] = fullFrame.data[(row*fullFrame.step) + column * 3 + 2];
			out.data[(row*out.step) + column * 3 + 1] = fullFrame.data[(row*fullFrame.step) + column * 3 + 1];
			out.data[(row*out.step) + column * 3] = fullFrame.data[(row*fullFrame.step) + column * 3];
		}
	}
}

__global__ void applyDilation(cv::cuda::GpuMat out, int *particleCount, int *alphas) {
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	int x = WINDOW_WIDTH;
	int y = WINDOW_HEIGHT;
	if (threadId < x * y)
	{
		int row = threadId / x;
		int column = threadId % x;
		if (particleCount[row * 5 * x + 5 * column + 4] > 0) {
			int particles = particleCount[row * 5 * x + 5 * column + 4];
			//particle count is sometimes not a valid number and it results in the pixel colours being above 255
			while (((particleCount[row * 5 * x + 5 * column] / particles) > 255) || ((particleCount[row * 5 * x + 5 * column + 1] / particles) > 255)||((particleCount[row * 5 * x + 5 * column + 2] / particles) > 255) || ((particleCount[row * 5 * x + 5 * column + 4] / particles) > 255)) {
				particles += 1;
			}
			for (int k = 0; k < 3; k++) {
				out.data[(row*out.step) + column * 3 + 2 - k] = particleCount[row * 5 * x + 5 * column + k] / particles;
			}
			alphas[row * x + column] = particleCount[row * 5 * x + 5 * column + 3] / particles;
		}	
		else {
			out.data[(row*out.step) + column * 3 + 2] = 0;
			out.data[(row*out.step) + column * 3 + 1] = 0;
			out.data[(row*out.step) + column * 3] = 0;
			alphas[row * x + column] = 0;
		}
	}
}

/*
flameFrame must start as a black frame
alpha transparency needs to be taken into account, and added to the smoke part
*/
__global__ void genericDilateKernel(cv::cuda::GpuMat flameFrame, int *particleCount, int *alphas)
{
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	int x = WINDOW_WIDTH;
	int y = WINDOW_HEIGHT;
	if (threadId < x * y)
	{
		int row = threadId / x;
		int column = threadId % x;
		uint8_t pixelR = flameFrame.data[(row*flameFrame.step) + column * 3 + 2];
		uint8_t pixelG = flameFrame.data[(row*flameFrame.step) + column * 3 + 1];
		uint8_t pixelB = flameFrame.data[(row*flameFrame.step) + column * 3];
		
		if ((pixelR + pixelG + pixelB) > 0)
		{
			for (int i = -6; i < 7; i++)
			//for (int i = -4; i < 5; i++)
			{
				for (int j = -6; j < 7; j++)
				//for (int j = -4; j < 5; j++)
				{
					if ((row + i > -1) && (row + i < y) && (column + j > -1) && (column + j < x))
					{
						particleCount[(row + i) * 5 * x + 5 * (column + j)] += pixelR;
						particleCount[(row + i) * 5 * x + 5 * (column + j) + 1] += pixelG;
						particleCount[(row + i) * 5 * x + 5 * (column + j) + 2] += pixelB;
						particleCount[(row + i) * 5 * x + 5 * (column + j) + 3] += alphas[row * x + column];
						//store count of particles used in this given pixel
						particleCount[(row + i) * 5 * x + 5 * (column + j) + 4] += 1;
					}
				}
			}
		}

	}
}

__host__ __device__
void Particle::setValues(float pos[3], float vel[3], unsigned char colour[4], float sizeI, float angleI, float weightI, float lifeI) {
	//position = pos;
	//velocity = vel;
	position[0] = pos[0];
	position[1] = pos[1];
	position[2] = pos[2];
	velocity[0] = vel[0];
	velocity[1] = vel[1];
	velocity[2] = vel[2];
	//acceleration = acc;
	r = colour[0];
	g = colour[1];
	b = colour[2];
	a = colour[3];
	//size may be irrelevant, if deeling with sub-pixel particles, however, this may result in a flame with gaps in
	size = sizeI;
	angle = angleI;
	weight = weightI;
	//particle life span must be longer as the particle's initial location approaches the match tracked location
	//life starts as a number (e.g. 4), and the particle is inactive when it's <= 0
	//all particles should be initialised to a life of 0
	life = lifeI;
}

//Update the state parameters of the particle based off it's acceleration, initial velocity and position
__host__ __device__
void Particle::updateParticle(float deltaT) {
	//update the values for this particle
	//postion =
	//velocity =
	//acceleration probably wont change, intiialise as a standard acceleration (in 3d, the acceleration must be for the 3d coordinates and then
	//appropriately converted to the 2D pixel coordinates)
	//acceleration is ignored, and instead just a fixed velocity could be used
	//acceleration =
	//life = 
	//Later, update colour based roughly off life span to emulate smoke or edge colouring of the fire
	//if the life is above the particle lifespan, then remove particle by resetting start inital particle attributes
}

__global__ void flameKernel(cv::cuda::GpuMat frame, Particle *container, int *alphas) {
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	int x = WINDOW_WIDTH;
	if (threadId < MaxParticles)
	{
		//printf("%d %d %f\n", MaxParticles, threadId, container[threadId].getLife());
		if ((threadId < MaxParticles) && (container[threadId].getLife() > 0)) {
			//int row = threadId / X;
			//int column = threadId % X;
			float *xyz = container[threadId].getPosition();
			int row = xyz[1];
			int column = xyz[0];
			//BGR pixel values
			if ((row >= 0) && (column >= 0) && (row < WINDOW_HEIGHT) && (column < WINDOW_WIDTH)) {
				frame.data[(row*frame.step) + column * 3] = container[threadId].getBlue();
				frame.data[(row*frame.step) + column * 3 + 1] = container[threadId].getGreen();
				frame.data[(row*frame.step) + column * 3 + 2] = container[threadId].getRed();
				alphas[row * x + column] = container[threadId].getAlpha();
			}
		}
	}
}

void addFlame(Mat frame, Mat fullFrame, Particle *container) {
	int x = WINDOW_WIDTH;
	int y = WINDOW_HEIGHT;
	int threadCount = 1024;
	int blocks = (MaxParticles - 1) / threadCount + 1;
	if (blocks == 1)
	{
		threadCount = MaxParticles;
	}
	Particle *d_container;
	//allocate device memory for the particle containe
	hipMalloc((void**)&d_container, sizeof(Particle) * MaxParticles);
	//transfer from host to device memory	
	hipMemcpy(d_container, container, sizeof(Particle) * MaxParticles, hipMemcpyHostToDevice);

	uint8_t *d_imgPtr;
	int *alphas;
	alphas = (int*)malloc(sizeof(int)*x*y);
	memset(alphas, 0, sizeof(int)*x*y);
	int *d_alphas;
	Mat newFrame(y, x, CV_8UC3, cv::Scalar(0, 0, 0));
	cv::cuda::GpuMat d_newFrame;
	//d_newFrame.upload(frame);
	d_newFrame.upload(newFrame);

	//Allocate device memory
	hipMalloc((void **)&d_imgPtr, d_newFrame.rows*d_newFrame.step);
	hipMalloc((void**)&d_alphas, sizeof(int)*x*y);
	hipMemcpyAsync(d_imgPtr, d_newFrame.ptr<uint8_t>(), d_newFrame.rows*d_newFrame.step, hipMemcpyDeviceToDevice);
	hipMemcpy(d_alphas, alphas, sizeof(int)*x*y, hipMemcpyHostToDevice);
	flameKernel << <blocks, threadCount >> > (d_newFrame, d_container, d_alphas);
	hipDeviceSynchronize();

	//free the device memory for the particle container
	hipFree(d_container);
	
	int *particleCount;
	particleCount = (int*)malloc(sizeof(int)*x*y*5);
	memset(particleCount, 0, sizeof(int)*x*y*5);
	uint8_t *d_fullFramePtr;
	cv::cuda::GpuMat d_fullFrame;
	
	d_fullFrame.upload(fullFrame);
	int *d_particleCount;
	hipMalloc((void**)&d_particleCount, sizeof(int)*x*y*5);
	hipMalloc((void**)&d_fullFramePtr, d_fullFrame.rows*d_fullFrame.step);
	hipMemcpy(d_particleCount, particleCount, sizeof(int)*x*y*5, hipMemcpyHostToDevice);
	hipMemcpyAsync(d_fullFramePtr, d_fullFrame.ptr<uint8_t>(), d_fullFrame.rows*d_fullFrame.step, hipMemcpyDeviceToDevice);
	

	threadCount = 1024;
	blocks = (x * y - 1) / threadCount + 1;
	if (blocks == 1)
	{
		threadCount = x * y;
	}

	genericDilateKernel << <blocks, threadCount >> > (d_newFrame, d_particleCount, d_alphas);
	hipDeviceSynchronize();

	//free new frame from device memory
	hipFree(d_imgPtr);
	d_newFrame.release();

	Mat out(y, x, CV_8UC3, cv::Scalar(0, 0, 0));
	//allocate flame frame to host and device memory
	uint8_t *d_flameFramePtr;
	cv::cuda::GpuMat d_flameFrame;
	d_flameFrame.upload(out);
	hipMalloc((void**)&d_flameFramePtr, d_fullFrame.rows*d_fullFrame.step);
	hipMemcpyAsync(d_flameFramePtr, d_flameFrame.ptr<uint8_t>(), d_flameFrame.rows*d_flameFrame.step, hipMemcpyDeviceToDevice);

	applyDilation << <blocks, threadCount >> > (d_flameFrame, d_particleCount, d_alphas);
	hipDeviceSynchronize();

	cv::cuda::GpuMat d_out;
	uint8_t *d_outPtr;
	d_out.upload(out);
	hipMalloc((void**)&d_outPtr, d_out.rows*d_out.step);
	hipMemcpyAsync(d_outPtr, d_out.ptr<uint8_t>(), d_out.rows*d_out.step, hipMemcpyDeviceToDevice);

	genericErodeKernel<<<blocks, threadCount>>>(d_out, d_flameFrame, d_fullFrame, d_alphas);
	hipDeviceSynchronize();
	d_out.download(frame);
	//Free frame pointers device memory	
	hipFree(d_outPtr);
	d_out.release();
	hipFree(d_fullFramePtr);
	d_fullFrame.release();
	hipFree(d_flameFramePtr);
	d_flameFrame.release();
	hipFree(d_particleCount);
	hipFree(d_alphas);

	//free host memory
	free(particleCount);
	free(alphas);


	hipError_t err = hipGetLastError();

	if (err != hipSuccess)
	{
		printf("CUDA Error: %s\n", hipGetErrorString(err));

		// Possibly: exit(-1) if program cannot continue....
	}
}

__global__ void particleKernel(Particle *container, int *matchTip, hiprandState_t *states){

	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadId < MaxParticles)
	{
		if (container[threadId].getLife() > 0) {
			//update all active particles
			//some may be reduced to a life below 0
			float *pos = container[threadId].getPosition();
			//float vel[3] = { 0.0, -300.0, 0.0 };
			float *vel = container[threadId].getVelocity();
			pos[1] += vel[1]*FrameTime;
			float size = 1;
			//angle and weight may be unnessecary for this particle system
			float angle = 0;
			float weight = 1;
			float life = container[threadId].getLife() + FrameTime;
			//unsigned char colour[4] = { container[threadId].getRed(), 85*log10f(32/life), life * 2 * 255, container[threadId].getAlpha() };
			unsigned char colour[4] = { container[threadId].getRed(), 85 * log10f(32 / life), container[threadId].getBlue(), 255};
			//give the particles a max life time
			if (life < 0.5){
				
				container[threadId].setValues(pos, vel, colour, size, angle, weight, life);
			}
			else if (life < 0.6) {
				//life*life*life*life*life may be faster than std::power(life, 5)
				unsigned char colour[4] = { 144, 144, 144, 8/(life*life*life*life*life)};
				container[threadId].setValues(pos, vel, colour, size, angle, weight, life);
			}
			else {
				//unsigned char colour[4] = { container[threadId].getRed(), 85 * log10f(32 / life), container[threadId].getBlue(), container[threadId].getAlpha() };
				unsigned char colour[4] = { 255, 255, 0, 255 };
				container[threadId].setValues(pos, vel, colour, size, angle, weight, 0);
			}
		}
		else if ((threadId < EmissionsPerFrame)&&(matchTip[0] > -1)) {
			//update EmissionsPerFrame particles that have a life <= 0
			//inactive particles, all will have a life <= 0
			//update these particles as new particles
			//it's possible this may be less than the number of emmissions per frame and that there still may be remaining inactive particles
			float width = 20;
			float baseVelocity = -200;
			hiprand_init(clock(), threadId, 0, &states[threadId]);
			float randomStartPosX = hiprand_uniform(&states[threadId])*width - (width/2) + float(matchTip[0]);
			float randomStartPosY = hiprand_uniform(&states[threadId])*(width/2) - (width / 4) + float(matchTip[1]);
			float velY = hiprand_uniform(&states[threadId])*200 + baseVelocity;
			if (velY > -50) {
				velY = -100;
			}
			float pos[3] = { randomStartPosX, randomStartPosY, 0.0 };
			//float vel[3] = { 0.0, -300.0, 0.0 };
			float vel[3] = { 0.0, velY, 0.0 };
			unsigned char colour[4] = { container[threadId].getRed(), container[threadId].getGreen(), container[threadId].getBlue(), container[threadId].getAlpha() };
			float size = 1;
			//angle and weight may be unnessecary for this particle system
			float angle = 0;
			float weight = 1;
			float life = FrameTime;
			container[threadId].setValues(pos, vel, colour, size, angle, weight, life);
		}
	}
}

__global__ void initialParticleKernel(Particle *container) {
	//printf("%d\n", MaxParticles);
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadId < MaxParticles)
	{
		float pos[3] = { 0.0, 0.0, 0.0 };
		float vel[3] = { 0.0, -300.0, 0.0 };
		unsigned char colour[4] = { 255, 255, 0, 255 };
		float size = 1;
		//angle and weight may be unnessecary for this particle system
		float angle = 0;
		float weight = 1;
		float life = 0;
		//printf("%f\n", life);
		container[threadId].setValues(pos, vel, colour, size, angle, weight, life);
	}
}

//update the particle postions and return the new positions, before adding the flame to the frame
//the particles are already sorted by their life, low to high
Particle *updateParticles(Particle *container, int matchTip[2]) {
	//add a new number of particles based on the emmissions per frame
	//max out at maxParticles
	//it's possible for particles to be removed, as they time out
	//we need a way to check if a particle is in use quickly so that particles can be overwritten
	//with new particles. The particle's age can act as this.
	//We can update a given number of known particles, so that they can be made visible
	//The particles would need to be sorted by age, or at the very least, guarenteed that the first
	//"emmisions per frame" particles are innactive
	//Alternatively, we could add another variable to the Particle class, a Boolean "Active", to indicate
	//whether the particle is active or not. This adds a little memory useage, but makes the sorting much
	//easier

	//reduced threadCount to fix the "Too many resources requested for launch" error
	int threadCount = 256;
	int blocks = (MaxParticles - 1) / threadCount + 1;
	if (blocks == 1)
	{
		threadCount = MaxParticles;
	}

	Particle *d_container;
	int *d_matchTip;
	hiprandState_t *d_randStates;
	//allocate device memory for the particle container aand match tip
	hipMalloc((void**)&d_container, sizeof(Particle) * MaxParticles);
	hipMalloc((void**)&d_matchTip, sizeof(int) * 2);
	hipMalloc((void**)&d_randStates, sizeof(hiprandState_t) * MaxParticles);
	//transfer from host to device memory	
	hipMemcpy(d_container, container, sizeof(Particle) * MaxParticles, hipMemcpyHostToDevice);
	hipMemcpy(d_matchTip, matchTip, sizeof(int) * 2, hipMemcpyHostToDevice);
	particleKernel<<<blocks, threadCount>>>(d_container, d_matchTip, d_randStates);
	

	hipDeviceSynchronize;
	hipMemcpy(container, d_container, sizeof(Particle) * MaxParticles, hipMemcpyDeviceToHost);
	hipFree(d_container);
	hipFree(d_matchTip);
	hipFree(d_randStates);
	//for debug only
	return container;
}

Particle *initialSetValues(Particle *container) {
	//pos, vel, colour, size, angle, weight, life
	//call a cuda kernel and initialise each of the particles simultaneously
	int threadCount = 1024;
	int blocks = (MaxParticles - 1) / threadCount + 1;
	if (blocks == 1)
	{
		threadCount = MaxParticles;
	}
	Particle *d_container;
	//allocate device memory for the particle container
	hipMalloc((void**)&d_container, sizeof(Particle) * MaxParticles);
	//transfer from host to device memory
	//hipMemcpy(d_container, container, sizeof(Particle) * MaxParticles, hipMemcpyHostToDevice);
	//allocate device memory for the maxParticle count
	//hipMalloc((void**)&d_maxParticles, sizeof(int));
	//transfer from host to device memory
	//hipMemcpy(d_maxParticles, maxParticles, sizeof(int), hipMemcpyHostToDevice);
	initialParticleKernel << <blocks, threadCount >> > (d_container);
	//hipError_t error = hipGetLastError();
	//if (error != hipSuccess){
	//	printf("%s\n", hipGetErrorString(error));
	//}
	hipDeviceSynchronize;
	//copy device memory for the particle container back to host memory
	hipMemcpy(container, d_container, sizeof(Particle) * MaxParticles, hipMemcpyDeviceToHost);
	//free(containerCopy);
	//free(maxParticlesCopy);
	hipFree(d_container);
	//for debug only
	return container;
}