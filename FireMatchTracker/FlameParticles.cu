#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include "matchTracker.h"
#include "Particle.h"

__host__ __device__
void Particle::setValues(float pos[3], float vel[3], unsigned char colour[4], float sizeI, float angleI, float weightI, float lifeI) {
	//position = pos;
	//velocity = vel;
	position[0] = pos[0];
	position[1] = pos[1];
	position[2] = pos[2];
	velocity[0] = vel[0];
	velocity[1] = vel[1];
	velocity[2] = vel[2];
	//acceleration = acc;
	r = colour[0];
	g = colour[1];
	b = colour[2];
	a = colour[3];
	//size may be irrelevant, if deeling with sub-pixel particles, however, this may result in a flame with gaps in
	size = sizeI;
	angle = angleI;
	weight = weightI;
	//particle life span must be longer as the particle's initial location approaches the match tracked location
	//life starts as a number (e.g. 4), and the particle is inactive when it's <= 0
	//all particles should be initialised to a life of 0
	life = lifeI;
}

//Update the state parameters of the particle based off it's acceleration, initial velocity and position
__host__ __device__
void Particle::updateParticle(float deltaT) {
	//update the values for this particle
	//postion =
	//velocity =
	//acceleration probably wont change, intiialise as a standard acceleration (in 3d, the acceleration must be for the 3d coordinates and then
	//appropriately converted to the 2D pixel coordinates)
	//acceleration is ignored, and instead just a fixed velocity could be used
	//acceleration =
	//life = 
	//Later, update colour based roughly off life span to emulate smoke or edge colouring of the fire
	//if the life is above the particle lifespan, then remove particle by resetting start inital particle attributes
}

Mat addFlame(Mat frame, int matchTip[2], Particle *container) {
	//for debug only
	return frame;
}

__global__ void particleKernel(Particle *container, int *matchTip){

	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadId < MaxParticles)
	{
		if (container[threadId].getLife() > 0) {
			//update all active particles
			//some may be reduced to a life below 0
			float life = container[0].getLife() + EmissionsPerFrame;
			//container[0].setValues(pos, vel, colour, size, angle, weight, life);
		}
		else if (threadId < EmissionsPerFrame) {
			//update EmissionsPerFrame particles that have a life <= 0
			//inactive particles, all will have a life <= 0
			//update these particles as new particles
			//it's possible this may be less than the number of emmissions per frame and that there still may be remaining inactive particles
			float life = container[0].getLife() + EmissionsPerFrame;

			//container[0].setValues(pos, vel, colour, size, angle, weight, life);
		}
	}
}

__global__ void initialParticleKernel(Particle *container) {
	//printf("%d\n", MaxParticles);
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadId < MaxParticles)
	{
		float pos[3] = { 0.0, 0.0, 0.0 };
		float vel[3] = { 0.0, 0.0, 0.0 };
		unsigned char colour[4] = { 0, 0, 0, 0 };
		float size = 1;
		//angle and weight may be unnessecary for this particle system
		float angle = 0;
		float weight = 1;
		float life = 0;
		//printf("%f\n", life);
		container[threadId].setValues(pos, vel, colour, size, angle, weight, life);
	}
}

//update the particle postions and return the new positions, before adding the flame to the frame
//the particles are already sorted by their life, low to high
Particle *updateParticles(Particle *container, int matchTip[2]) {
	//add a new number of particles based on the emmissions per frame
	//max out at maxParticles
	//it's possible for particles to be removed, as they time out
	//we need a way to check if a particle is in use quickly so that particles can be overwritten
	//with new particles. The particle's age can act as this.
	//We can update a given number of known particles, so that they can be made visible
	//The particles would need to be sorted by age, or at the very least, guarenteed that the first
	//"emmisions per frame" particles are innactive
	//Alternatively, we could add another variable to the Particle class, a Boolean "Active", to indicate
	//whether the particle is active or not. This adds a little memory useage, but makes the sorting much
	//easier
	float pos[3] = { 0.0, 0.0, 0.0 };
	float vel[3] = { 0.0, 0.0, 0.0 };
	unsigned char colour[4] = { 0, 0, 0, 0 };
	float size = 1;
	//angle and weight may be unnessecary for this particle system
	float angle = 0;
	float weight = 1;

	int threadCount = 1024;
	int blocks = (MaxParticles - 1) / threadCount + 1;
	if (blocks == 1)
	{
		threadCount = MaxParticles;
	}

	Particle *d_container;
	int *d_matchTip;
	//allocate device memory for deltaT, the particle container and the emissions per frame
	hipMalloc((void**)&d_container, sizeof(Particle) * MaxParticles);
	hipMalloc((void**)&d_matchTip, sizeof(int) * 2);
	//transfer from host to device memory	
	hipMemcpy(d_container, container, sizeof(Particle) * MaxParticles, hipMemcpyHostToDevice);
	hipMemcpy(d_matchTip, matchTip, sizeof(int) * 2, hipMemcpyHostToDevice);
	particleKernel<<<blocks, threadCount>>>(d_container, d_matchTip);
	
	hipFree(d_container);
	hipFree(d_matchTip);
	//for debug only
	return container;
}

Particle *initialSetValues(Particle *container) {
	//pos, vel, colour, size, angle, weight, life
	//call a cuda kernel and initialise each of the particles simultaneously
	int threadCount = 1024;
	int blocks = (MaxParticles - 1) / threadCount + 1;
	if (blocks == 1)
	{
		threadCount = MaxParticles;
	}
	Particle *d_container;
	//allocate device memory for the particle container
	hipMalloc((void**)&d_container, sizeof(Particle) * MaxParticles);
	//transfer from host to device memory
	//hipMemcpy(d_container, container, sizeof(Particle) * MaxParticles, hipMemcpyHostToDevice);
	//allocate device memory for the maxParticle count
	//hipMalloc((void**)&d_maxParticles, sizeof(int));
	//transfer from host to device memory
	//hipMemcpy(d_maxParticles, maxParticles, sizeof(int), hipMemcpyHostToDevice);
	initialParticleKernel << <blocks, threadCount >> > (d_container);
	//hipError_t error = hipGetLastError();
	//if (error != hipSuccess){
	//	printf("%s\n", hipGetErrorString(error));
	//}
	hipDeviceSynchronize;
	//copy device memory for the particle container back to host memory
	hipMemcpy(container, d_container, sizeof(Particle) * MaxParticles, hipMemcpyDeviceToHost);
	//free(containerCopy);
	//free(maxParticlesCopy);
	hipFree(d_container);
	//for debug only
	return container;
}