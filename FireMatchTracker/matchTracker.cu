#include "hip/hip_runtime.h"
#include "matchTracker.h"
#include <set>
using namespace cv::cuda;

/*
Detect the centre of red objects and mark these as pixels on a black frame
trackedFrame - A black frame with single red pixels representing the centre of red detected objects
cleanFrame - A frame of red and black, where red represents particularly red parts of the image
*/
__global__ void detectObjectKernel(cv::cuda::GpuMat trackedFrame, cv::cuda::GpuMat cleanFrame)
{
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	if ((threadId < X * Y) && ((threadId/X)%10 == 0) && ((threadId % X) % 10 == 0))
	{
		int row = threadId / X;
		int column = threadId % X;
		uint8_t pixelGClean = cleanFrame.data[(row*cleanFrame.step) + column * 3 + 1];
		if (pixelGClean == 255) {
			int maxX = column;
			int maxY = row;
			int minX = column;
			int minY = row;
			//Current array size is unstable, only works up until half the pixels being red (displayed as green), but saves memory and
			//it's expected there's effort to be as few red pixels as possible
			int pixelList[(X * Y) / 200];
			bool pixelUsed[X / 10][Y / 10] = { 0 };
			pixelList[0] = X * row + column;
			int listLength = 1;
			uint8_t pixelGcleanTest;
			int x, y, xTemp, yTemp, temp;
			while (listLength > 0) {
				x = pixelList[0] % X;
				y = pixelList[0] / X;
				if (!pixelUsed[x / 10][y / 10]) {
					if (x < X - 10) {
						pixelGcleanTest = cleanFrame.data[y*cleanFrame.step + (x + 10) * 3 + 1];
						if ((pixelGcleanTest == 255) && (!pixelUsed[(x + 10) / 10][y / 10])) {
							pixelList[listLength] = X * y + x + 10;
							listLength++;
							if (x + 10 > maxX) {
								maxX = maxX + 10;
							}
						}
					}
					if (x > 9) {
						pixelGcleanTest = cleanFrame.data[y*cleanFrame.step + (x - 10) * 3 + 1];
						if ((pixelGcleanTest == 255) && (!pixelUsed[(x - 10) / 10][y / 10])) {
							pixelList[listLength] = X * y + x - 10;
							listLength++;
							if (x - 10 < minX) {
								minX = minX - 10;
							}
						}
					}
					if (y < Y - 10) {
						pixelGcleanTest = cleanFrame.data[(y + 10)*cleanFrame.step + x * 3 + 1];
						if ((pixelGcleanTest == 255) && (!pixelUsed[x / 10][(y + 10) / 10])) {
							pixelList[listLength] = X * (y + 10) + x;
							listLength++;
							if (y + 10 > maxY) {
								maxY = maxY + 10;
							}
						}
					}
					if (y > 9) {
						pixelGcleanTest = cleanFrame.data[(y - 10)*cleanFrame.step + x * 3 + 1];
						if ((pixelGcleanTest == 255) && (!pixelUsed[x / 10][(y - 10) / 10])) {
							pixelList[listLength] = X * (y - 10) + x;
							listLength++;
							if (y - 10 < minY) {
								minY = minY - 10;
							}
						}
					}
				}
				pixelUsed[x / 10][y / 10] = true;
				temp = pixelList[listLength - 1];
				pixelList[listLength - 1] = pixelList[0];
				pixelList[0] = temp;
				listLength--;
			}
			int centreX = (maxX - minX) / 2 + minX;
			int centreY = (maxY - minY) / 2 + minY;
			trackedFrame.data[(centreY*trackedFrame.step) + centreX * 3 + 2] = 255;
		}
	}
}

/*
Erode the red image to restore closer to original object sizes
out - output of the eroded frame, red objects on a black background
dilatedFrame - input of an already dilated frame, red objects on a black background
*/
__global__ void erodeKernel(cv::cuda::GpuMat out, cv::cuda::GpuMat dilatedFrame)
{
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadId < X * Y)
	{
		int row = threadId / X;
		int column = threadId % X;
		uint8_t pixelR = dilatedFrame.data[(row*dilatedFrame.step) + column * 3 + 2];

		if (pixelR == 255)
		{
			bool allPixelsRed = true;
			for (int i = -2; i < 3; i++)
			{
				for (int j = -2; j < 3; j++)
				{
					if ((row + i > -1) && (row + i < Y) && (column + j > -1) && (column + j < X))
					{
						if (dilatedFrame.data[((row + i)*dilatedFrame.step) + (column + j) * 3 + 2] == 0)
						{
							allPixelsRed = false;
						}
					}
				}
			}
			if (allPixelsRed)
			{
				out.data[(row*out.step) + column * 3] = 0;
				out.data[(row*out.step) + column * 3 + 1] = 255;
				out.data[(row*out.step) + column * 3 + 2] = 0;
			}
		}
	}
}

/*
Dilate the red object on black background frame, to fill gaps inside the objects, if any are present
out - out of the dilated frame, in the form of red objects on a black background
redFrame - a frame with all of the detected red areas of the image highlighted, on a black background
*/
__global__ void dilateKernel(cv::cuda::GpuMat out, cv::cuda::GpuMat redFrame)
{
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadId < X * Y)
	{
		int row = threadId / X;
		int column = threadId % X;
		uint8_t pixelR = redFrame.data[(row*redFrame.step) + column * 3 + 2];

		if (pixelR == 255)
		{
			for (int i = -6; i < 7; i++)
			{
				for (int j = -6; j < 7; j++)
				{
					if (!(i == 0 && j == 0))
					{
						if ((row + i > -1) && (row + i < Y) && (column + j > -1) && (column + j < X))
						{
							out.data[((row + i)*out.step) + (column + j) * 3 + 2] = 255;
						}
					}
				}
			}
		}		
	}
}

/*
Identify the areas of an image that are particularly red, so that the markers on the match can be detected
out - output, a frame with all of the areas that pass a "redness" threshold as red and the rest as black
*/
__global__ void getRedKernel(cv::cuda::GpuMat out)
{
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadId < X * Y)
	{
		int row = threadId / X;
		int column = threadId % X;
		//BGR pixel values
		uint8_t pixelB = out.data[(row*out.step) + column * 3];
		uint8_t pixelG = out.data[(row*out.step) + column * 3 + 1];
		uint8_t pixelR = out.data[(row*out.step) + column * 3 + 2];
		if ((pixelR > 80) && (pixelB < 10) && (pixelG < 10))
		{
			out.data[(row*out.step) + column * 3] = 0;
			out.data[(row*out.step) + column * 3 + 1] = 0;
			out.data[(row*out.step) + column * 3 + 2] = 255;
		}
		else
		{
			out.data[(row*out.step) + column * 3] = 0;
			out.data[(row*out.step) + column * 3 + 1] = 0;
			out.data[(row*out.step) + column * 3 + 2] = 0;
		}
	}
}

/*
Create a black mat image
out - an entirely black frame
*/
__global__ void blackKernel(cv::cuda::GpuMat out)
{
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadId < X * Y)
	{
		int row = threadId / X;
		int column = threadId % X;
		out.data[(row*out.step) + column * 3] = 0;
		out.data[(row*out.step) + column * 3 + 1] = 0;
		out.data[(row*out.step) + column * 3 + 2] = 0;
	}
}

/*
Identify where the match tip is in the image, based on the tracking dot centres of red areas and their relationship with each
other. The expectation is that there must be three tracking locations in a straight line, where the ratio between 2 locations
compared to another 1 with a shared location is approximately 1.5.
trackingLocations - The locations of the red tracking dots in the frame, that represent the center of red areas
matchTip - The output, an x-y coordinate for the determined match tip in the frame
*/
void getMatchLocation(std::set<int> *trackingLocations, int *matchTip){
	matchTip[0] = -1;
	matchTip[1] = -1;
	int trackA[2];
	int trackB[2];
	int trackC[2];
	int a[2];
	int b[2];
	double aMagnitude;
	double bMagnitude;
	double ratio;
	double oldRatio = 0.0;
	double dotProduct;
	//iterate over set to find all 3 location combinations, and find the most likely one to be the matchstick
	for (auto i : *trackingLocations) {
		for (auto j : *trackingLocations) {
			if (i != j) {
				for (auto k : *trackingLocations) {
					if ((i != k) && (j != k)) {
						trackA[0] = i % X;
						trackA[1] = i / X;
						trackB[0] = j % X;
						trackB[1] = j / X;
						trackC[0] = k % X;
						trackC[1] = k / X;
						a[0] = trackB[0] - trackA[0];
						a[1] = trackB[1] - trackA[1];
						b[0] = trackC[0] - trackB[0];
						b[1] = trackC[1] - trackB[1];
						aMagnitude = sqrt(a[0] * a[0] + a[1] * a[1]);
						bMagnitude = sqrt(b[0] * b[0] + b[1] * b[1]);
						dotProduct = (a[0] * b[0] + a[1] * b[1]) / (aMagnitude * bMagnitude);	
						if (dotProduct > 0.99) {
							ratio = bMagnitude / aMagnitude;
							//test if ratio is close to 1.5, as this is approximately the expected ratio
							if ((1.35 < ratio) && (ratio < 1.65) && (abs(1.6 - ratio) < abs(1.6 - oldRatio))){
								memcpy(matchTip, trackC, sizeof(int)*2);
								matchTip[0] += b[0] / 10;
								matchTip[1] += b[1] / 10;
								oldRatio = ratio;
							}
						}
					}
				}
			}			
		}
	}
}

/*
The main function for tracking the tip of the match. Take an input frame from the webcam and find the location where flame particles
should be emitted from, as an x-y coordinate of the match tip.
frame - The frame taken from the webcam
tip - The pointer to the memory location to store the tip coordinates
*/
void track(Mat frame, int *tip) {
	int threadCount = 1024;
	int blocks = (X * Y - 1) / threadCount + 1;
	if (blocks == 1)
	{
		threadCount = X * Y;
	}
	//Set up device variables
	uint8_t *d_outPtr;
	cv::cuda::GpuMat d_outFrame;
	d_outFrame.upload(frame);

	//Allocate device memory
	hipMalloc((void **)&d_outPtr, d_outFrame.rows*d_outFrame.step);
	hipMemcpyAsync(d_outPtr, d_outFrame.ptr<uint8_t>(), d_outFrame.rows*d_outFrame.step, hipMemcpyDeviceToDevice);
	getRedKernel << <blocks, threadCount >> > (d_outFrame);

	uint8_t *d_dilatedPtr;
	cv::cuda::GpuMat d_dilatedFrame;
	d_outFrame.copyTo(d_dilatedFrame);

	//Allocate new device memory
	hipMalloc((void**)&d_dilatedPtr, d_dilatedFrame.rows*d_dilatedFrame.step);
	hipMemcpyAsync(d_dilatedPtr, d_dilatedFrame.ptr<uint8_t>(), d_dilatedFrame.rows*d_dilatedFrame.step, hipMemcpyDeviceToDevice);

	dilateKernel << <blocks, threadCount >> > (d_dilatedFrame, d_outFrame);
	//Free outFrame pointer device memory
	hipFree(d_outPtr);
	d_outFrame.release();

	uint8_t *d_erodedPtr;
	cv::cuda::GpuMat d_erodedFrame;
	d_dilatedFrame.copyTo(d_erodedFrame);

	//Allocated new device memory
	hipMalloc((void**)&d_erodedPtr, d_erodedFrame.rows*d_erodedFrame.step);
	hipMemcpyAsync(d_erodedPtr, d_erodedFrame.ptr<uint8_t>(), d_erodedFrame.rows*d_erodedFrame.step, hipMemcpyDeviceToDevice);

	//convert the frame to be completely black to avoid weird artifacts
	blackKernel << <blocks, threadCount >> > (d_erodedFrame);
	erodeKernel << <blocks, threadCount >> > (d_erodedFrame, d_dilatedFrame);
	//Free dilatedFrame pointer device memory
	hipFree(d_dilatedPtr);
	d_dilatedFrame.release();

	uint8_t *d_trackedFramePtr;
	cv::cuda::GpuMat d_trackedFrame;
	//declare a totally black image matrix
	Mat newFrame(Y, X, CV_8UC3, cv::Scalar(0, 0, 0));
	d_trackedFrame.upload(newFrame);
	//Allocate new device memory
	hipMalloc((void**)&d_trackedFramePtr, d_trackedFrame.rows*d_trackedFrame.step);
	hipMemcpyAsync(d_trackedFramePtr, d_trackedFrame.ptr<uint8_t>(), d_trackedFrame.rows*d_trackedFrame.step, hipMemcpyDeviceToDevice);
	
	detectObjectKernel<<<blocks, threadCount>>>(d_trackedFrame, d_erodedFrame);

	//Free erodedFrame pointer device memory
	hipFree(d_erodedPtr);
	d_erodedFrame.release();

	hipError_t error2 = hipGetLastError();
	if (error2 != hipSuccess) {
		printf("2. Error: %s\n", hipGetErrorString(error2));
	}
	Mat trackedFrame;
	d_trackedFrame.download(trackedFrame);
	std::set<int> *trackingLocations = new std::set<int>[(X / 20)*(Y / 20)];
	for (int i = 0; i < Y; i = i + 5) {
		for (int j = 0; j < X; j = j + 5) {
			if (trackedFrame.data[(i*trackedFrame.step) + j * 3 + 2] == 255) {
				int thisPixel = i * X + j;
				trackingLocations[0].insert(thisPixel);
			}
		}
	}

	//Free the tracked frame from device memory
	hipFree(d_trackedFramePtr);
	d_trackedFrame.release();
	trackedFrame.release();

	getMatchLocation(trackingLocations, tip);
	//uncomment for adding tracking marker to the frame
	/*
	if ((tip[0] > 0) && (tip[0] < X - 1) && (tip[0] > 0) && (tip[1] < Y - 1) && (tip[1] > 0)) {
		frame.data[tip[1] * frame.step + tip[0] * 3] = 0;
		frame.data[tip[1] * frame.step + tip[0] * 3 + 1] = 255;
		frame.data[tip[1] * frame.step + tip[0] * 3 + 2] = 0;
		frame.data[(tip[1] + 1) * frame.step + tip[0] * 3] = 0;
		frame.data[(tip[1] + 1) * frame.step + tip[0] * 3 + 1] = 255;
		frame.data[(tip[1] + 1) * frame.step + tip[0] * 3 + 2] = 0;
		frame.data[(tip[1] - 1) * frame.step + tip[0] * 3] = 0;
		frame.data[(tip[1] - 1) * frame.step + tip[0] * 3 + 1] = 255;
		frame.data[(tip[1] - 1) * frame.step + tip[0] * 3 + 2] = 0;
		frame.data[tip[1] * frame.step + (tip[0] + 1) * 3] = 0;
		frame.data[tip[1] * frame.step + (tip[0] + 1) * 3 + 1] = 255;
		frame.data[tip[1] * frame.step + (tip[0] + 1) * 3 + 2] = 0;
		frame.data[(tip[1] + 1) * frame.step + (tip[0] + 1) * 3] = 0;
		frame.data[(tip[1] + 1) * frame.step + (tip[0] + 1) * 3 + 1] = 255;
		frame.data[(tip[1] + 1) * frame.step + (tip[0] + 1) * 3 + 2] = 0;
		frame.data[(tip[1] - 1) * frame.step + (tip[0] + 1) * 3] = 0;
		frame.data[(tip[1] - 1) * frame.step + (tip[0] + 1) * 3 + 1] = 255;
		frame.data[(tip[1] - 1) * frame.step + (tip[0] + 1) * 3 + 2] = 0;
		frame.data[tip[1] * frame.step + (tip[0] - 1) * 3] = 0;
		frame.data[tip[1] * frame.step + (tip[0] - 1) * 3 + 1] = 255;
		frame.data[tip[1] * frame.step + (tip[0] - 1) * 3 + 2] = 0;
		frame.data[(tip[1] + 1) * frame.step + (tip[0] - 1) * 3] = 0;
		frame.data[(tip[1] + 1) * frame.step + (tip[0] - 1) * 3 + 1] = 255;
		frame.data[(tip[1] + 1) * frame.step + (tip[0] - 1) * 3 + 2] = 0;
		frame.data[(tip[1] - 1) * frame.step + (tip[0] - 1) * 3] = 0;
		frame.data[(tip[1] - 1) * frame.step + (tip[0] - 1) * 3 + 1] = 255;
		frame.data[(tip[1] - 1) * frame.step + (tip[0] - 1) * 3 + 2] = 0;
	}
	*/
	delete [] trackingLocations;
}