#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <iostream>
using namespace std;
#include <opencv2/core.hpp>
using namespace cv;
#include <opencv2/cudaarithm.hpp>
using namespace cv::cuda;

__global__ void trackKernel(int *c, const int *a, const int *b)
{
	//detect end
	//determine orientation
	//determine distance
	//have internal representation of it's position in 3D
	//draw particles in 3D space
	//move particles with physics based on the match's movement
}

int main()
{
	//test code from Get started with OpenCV CUDA cpp
	printShortCudaDeviceInfo(getDevice());
	int cuda_devices_number = getCudaEnabledDeviceCount();
	cout << "CUDA Device(s) Number: " << cuda_devices_number << endl;
	DeviceInfo _deviceInfo;
	bool _isd_evice_compatible = _deviceInfo.isCompatible();
	cout << "CUDA Device(s) Compatible: " << _isd_evice_compatible << endl;
	return 0;
}