#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include "matchTracker.h"
#include <opencv2/cudaarithm.hpp>
#include <opencv2/cudafilters.hpp>
#include "opencv2/cudaimgproc.hpp"
using namespace cv::cuda;


#define X 1280
#define Y 720

//Not ideal, expectedly produces motion blur, but not in a nice way
__global__ void averageKernel(cv::cuda::GpuMat out, cv::cuda::PtrStepSz<uint8_t[3]> bufferFrames[3])
{
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadId < X * Y)
	{
		int row = threadId / X;
		int column = threadId % X;
		bool colour = false;
		for (int i = 0; i < 3; i++) {
			//printf("1. %d\n", &bufferFrames[i]);
			//cv::cuda::GpuMat thisMat = (cv::cuda::GpuMat) *bufferFrames[i];
			//uint8_t B = bufferFrames[i](row, column)[0];
			uint8_t G = bufferFrames[i](row, column)[1];
			//uint8_t R = bufferFrames[i](row, column)[2];
			//if ((B > 0)||(G > 0)||(R > 0)) {
			//	printf("%d %d %d\n", B, G, R);
			//}

			if (G > 0) {
				//printf("%d %d\n", row, column);
				colour = true;
			}
			
			//printf("%d %d %d\n", row, column, bufferFrames[i](row, column));
			//if (bufferFrames[i].ptr(row)[column] > 0) {
				//printf("%d %d %f\n", row, column, bufferFrames[i].ptr(row)[column]);
			//}			
			//if (bufferFrames[i].data[(row*bufferFrames[i].step) + column * 3 + 1] > 0) {
			//	colour = true;
			//	printf(":D\n");
			//}
		}
		

		if (colour)
		{
			out.data[(row*out.step) + column * 3] = 0;
			out.data[(row*out.step) + column * 3 + 1] = 0;
			out.data[(row*out.step) + column * 3 + 2] = 255;
		}
	}
}

__global__ void detectObjectKernel(uint8_t *a, cv::cuda::GpuMat cleanFrame, cv::cuda::GpuMat frameCopy)
{
	//detect object size here
	printf("test\n");
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadId < X * Y)
	{
		int row = threadId / X;
		int column = threadId % X;
		printf("%d", cleanFrame.data[(row*cleanFrame.step) + column * 3 + 1]);
		if (cleanFrame.data[(row*cleanFrame.step) + column * 3 + 1] > 0) {
			int maxX = column;
			int maxY = row;
			int minX = column;
			int minY = row;
			bool traversable = true;
			int pixelList[2][X*Y];
			pixelList[0][0] = column;
			pixelList[1][0] = row;
			int listLength = 1;
			frameCopy.data[row*frameCopy.step + column * 3 + 1] = 0;
			while (traversable){
				int newPixels = 0;
				for (int i = 0; i < listLength; i++) {
					int x = pixelList[0][i];
					int y = pixelList[1][i];
					if (x < X){
						if (frameCopy.data[y*frameCopy.step + (x + 1) * 3 + 1] > 0) {
							pixelList[0][listLength] = x + 1;
							pixelList[1][listLength] = y;
							listLength++;
							newPixels++;
							frameCopy.data[y*frameCopy.step + (x + 1) * 3 + 1] = 0;
							maxX++;
						}
					}
					if (x > 0){
						if (frameCopy.data[y*frameCopy.step + (x - 1) * 3 + 1] > 0) {
							pixelList[0][listLength] = x - 1;
							pixelList[1][listLength] = y;
							listLength++;
							newPixels++;
							frameCopy.data[y*frameCopy.step + (x - 1) * 3 + 1] = 0;
							minX--;
						}
					}
					if (y < Y){
						if (frameCopy.data[(y + 1)*frameCopy.step + x * 3 + 1] > 0) {
							pixelList[0][listLength] = x;
							pixelList[1][listLength] = y + 1;
							listLength++;
							newPixels++;
							frameCopy.data[(y + 1)*frameCopy.step + x * 3 + 1] = 0;
							maxY++;
						}
					}
					if (y > 0){
						if (frameCopy.data[(y - 1)*frameCopy.step + x * 3 + 1] > 0) {
							pixelList[0][listLength] = x;
							pixelList[1][listLength] = y - 1;
							listLength++;
							newPixels++;
							frameCopy.data[(y - 1)*frameCopy.step + x * 3 + 1] = 0;
							minY--;
						}
					}
				}
				if (newPixels == 0) {
					traversable = false;
					printf("%d %d %d %d\n", minX, maxX, minY, maxY);
				}
			}
		}
	}
}

__global__ void erodeKernel(cv::cuda::GpuMat out, cv::cuda::GpuMat dilatedFrame)
{
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadId < X * Y)
	{
		int row = threadId / X;
		int column = threadId % X;
		uint8_t pixelR = dilatedFrame.data[(row*dilatedFrame.step) + column * 3 + 2];

		if (pixelR == 255)
		{
			bool allPixelsRed = true;
			//for (int i = -5; i < 6; i++)
			//for (int i = -8; i < 9; i++)
			for (int i = -2; i < 3; i++)
			{
				//for (int j = -5; j < 6; j++)
				//for (int j = -8; j < 9; j++)
				for (int j = -2; j < 3; j++)
				{
					if ((row + i > -1) && (row + i < Y) && (column + j > -1) && (column + j < X))
					{
						if (dilatedFrame.data[((row + i)*dilatedFrame.step) + (column + j) * 3 + 2] == 0)
						{
							allPixelsRed = false;
						}
					}
				}
			}
			if (allPixelsRed)
			{
				out.data[(row*out.step) + column * 3] = 0;
				out.data[(row*out.step) + column * 3 + 1] = 255;
				out.data[(row*out.step) + column * 3 + 2] = 0;
			}
		}
	}
}

__global__ void dilateKernel(cv::cuda::GpuMat out, cv::cuda::GpuMat redFrame)
{
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadId < X * Y)
	{
		int row = threadId / X;
		int column = threadId % X;
		uint8_t pixelR = redFrame.data[(row*redFrame.step) + column * 3 + 2];

		if (pixelR == 255)
		{
			//for (int i = -8; i < 9; i++)
			for (int i = -6; i < 7; i++)
			{
				for (int j = -6; j < 7; j++)
				{
					if (!(i == 0 && j == 0))
					{
						if ((row + i > -1) && (row + i < Y) && (column + j > -1) && (column + j < X))
						{
							out.data[((row + i)*out.step) + (column + j) * 3 + 2] = 255;
						}
					}
				}
			}
		}		
	}
}

__global__ void getRedKernel(cv::cuda::GpuMat out, cv::cuda::GpuMat frame)
{
	//detect end
	//determine orientation
	//determine distance
	//have internal representation of it's position in 3D
	//draw particles in 3D space
	//move particles with physics based on the match's movement
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadId < X * Y)
	{
		int row = threadId / X;
		int column = threadId % X;
		//BGR pixel values
		uint8_t pixelB = frame.data[(row*frame.step) + column * 3];
		uint8_t pixelG = frame.data[(row*frame.step) + column * 3 + 1];
		uint8_t pixelR = frame.data[(row*frame.step) + column * 3 + 2];
		//out.data[(row*out.step) + column * 3] = pixelB;
		//out.data[(row*out.step) + column * 3 + 1] = pixelG;
		//out.data[(row*out.step) + column * 3 + 2] = pixelR;
		//if ((pixelR > 128) && (pixelB < 50) && (pixelG < 50))
		if ((pixelR > 80) && (pixelB < 10) && (pixelG < 10))
		//if (((pixelR > 128) && (pixelB < 10) && (pixelG < 10))||((pixelR > 100)&&(pixelB < 4)&&(pixelG < 4)) || ((pixelR > 90) && (pixelB < 1) && (pixelG < 1)))
		//if (((pixelR > 100) && (pixelB < 5) && (pixelG < 5)))
		//if ((pixelR > 4*(pixelB + pixelG))&&(pixelR > 110))
		{

			//out.data[(row*out.step) + column * 3] = pixelB;
			//out.data[(row*out.step) + column * 3 + 1] = pixelG;
			//out.data[(row*out.step) + column * 3 + 2] = pixelR;
			out.data[(row*out.step) + column * 3] = 0;
			out.data[(row*out.step) + column * 3 + 1] = 0;
			out.data[(row*out.step) + column * 3 + 2] = 255;
		}
		else
		{
			//out.data[(row*out.step) + column * 3] = pixelB/2;
			//out.data[(row*out.step) + column * 3 + 1] = pixelG/2;
			//out.data[(row*out.step) + column * 3 + 2] = pixelR/2;
			out.data[(row*out.step) + column * 3] = 0;
			out.data[(row*out.step) + column * 3 + 1] = 0;
			out.data[(row*out.step) + column * 3 + 2] = 0;
		}
	}
}

__global__ void blackKernel(cv::cuda::GpuMat out)
{
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadId < X * Y)
	{
		int row = threadId / X;
		int column = threadId % X;
		out.data[(row*out.step) + column * 3] = 0;
		out.data[(row*out.step) + column * 3 + 1] = 0;
		out.data[(row*out.step) + column * 3 + 2] = 0;
	}
}

Mat track(Mat frame) {
	//Mat *newFrame = (Mat*)malloc(X * Y * sizeof(Mat));
	//Mat *newFrame = (Mat*)malloc(sizeof(frame));
	//Mat *outFrame = (Mat*)malloc(sizeof(frame));
	//*newFrame = frame.clone();
	int threadCount = 1024;
	int blocks = (X * Y - 1) / threadCount + 1;
	if (blocks == 1)
	{
		threadCount = X * Y;
	}
	//Set up device variables
	//Mat *d_newFrame;
	//Mat *d_outFrame;
	uint8_t *d_imgPtr;
	uint8_t *d_outPtr;
	cv::cuda::GpuMat d_newFrame;
	cv::cuda::GpuMat d_outFrame;
	d_newFrame.upload(frame);
	d_outFrame.upload(frame);

	//Allocate device memory
	hipMalloc((void **)&d_imgPtr, d_newFrame.rows*d_newFrame.step);
	hipMalloc((void **)&d_outPtr, d_outFrame.rows*d_outFrame.step);
	hipMemcpyAsync(d_imgPtr, d_newFrame.ptr<uint8_t>(), d_newFrame.rows*d_newFrame.step, hipMemcpyDeviceToDevice);
	hipMemcpyAsync(d_outPtr, d_outFrame.ptr<uint8_t>(), d_outFrame.rows*d_outFrame.step, hipMemcpyDeviceToDevice);
	//hipMalloc((void**)&d_newFrame, sizeof(frame));
	//hipMalloc((void**)&d_outFrame, sizeof(frame));
	//transfer memory from host to device memory
	//hipMemcpy(d_newFrame, newFrame, sizeof(frame), hipMemcpyHostToDevice);
	getRedKernel<<<blocks, threadCount>>>(d_outFrame, d_newFrame);
	//Free newFrame device and host memory
	//hipFree(d_newFrame);
	//free(newFrame);
	//hipMemcpy(outFrame, d_outFrame, sizeof(frame), hipMemcpyDeviceToHost);
	//Image dilation
	//int erosionDilation_size = 5;
	//Mat element = cv::getStructuringElement(MORPH_RECT, Size(2 * erosionDilation_size + 1, 2 * erosionDilation_size + 1));
	//Ptr<cuda::Filter> dilateFilter = cv::cuda::createMorphologyFilter(MORPH_DILATE, d_outFrame.type(), element);
	//dilateFilter->apply(d_outFrame, d_outFrame);
	
	//Free original frame pointer device memory
	hipFree(d_imgPtr);
	d_newFrame.release();
	

	uint8_t *d_dilatedPtr;
	cv::cuda::GpuMat d_dilatedFrame;
	d_outFrame.copyTo(d_dilatedFrame);
	

	
	//Allocate new device memory
	hipMalloc((void**)&d_dilatedPtr, d_dilatedFrame.rows*d_dilatedFrame.step);
	hipMemcpyAsync(d_dilatedPtr, d_dilatedFrame.ptr<uint8_t>(), d_dilatedFrame.rows*d_dilatedFrame.step, hipMemcpyDeviceToDevice);

	dilateKernel<<<blocks, threadCount>>>(d_dilatedFrame, d_outFrame);	
	
	//Free outFrame pointer device memory
	hipFree(d_outPtr);
	d_outFrame.release();

	uint8_t *d_erodedPtr;
	cv::cuda::GpuMat d_erodedFrame;
	d_dilatedFrame.copyTo(d_erodedFrame);

	//Allocated new device memory
	hipMalloc((void**)&d_erodedPtr, d_erodedFrame.rows*d_erodedFrame.step);
	hipMemcpyAsync(d_erodedPtr, d_erodedFrame.ptr<uint8_t>(), d_erodedFrame.rows*d_erodedFrame.step, hipMemcpyDeviceToDevice);

	//convert the frame to be completely black to avoid weird artifacts
	blackKernel<<<blocks, threadCount>>>(d_erodedFrame);
	
	erodeKernel<<<blocks, threadCount>>>(d_erodedFrame, d_dilatedFrame);
	
	//Free dilatedFrame pointer device memory
	hipFree(d_dilatedPtr);
	d_dilatedFrame.release();

	uint8_t *trackingLocations = (uint8_t*)malloc(3 * 100 * sizeof(uint8_t));
	uint8_t *d_trackingLocations;
	uint8_t *d_copyFramePtr;
	cv::cuda::GpuMat d_copyFrame;
	d_erodedFrame.copyTo(d_copyFrame);
	//Allocate new device memory
	hipMalloc((void**)&d_copyFramePtr, d_copyFrame.rows*d_copyFrame.step);
	hipMemcpyAsync(d_copyFramePtr, d_copyFrame.ptr<uint8_t>(), d_copyFrame.rows*d_copyFrame.step, hipMemcpyDeviceToDevice);

	hipMalloc((void**)&d_trackingLocations, sizeof(uint8_t) * 3 * 100);
	detectObjectKernel<<<blocks, threadCount>>>(d_trackingLocations, d_erodedFrame, d_copyFrame);

	//preventing memory leaks, in the wrong positon right now, purposely
	free(trackingLocations);
	hipFree(d_trackingLocations);
	hipFree(d_copyFramePtr);


	Mat outFrame;
	d_erodedFrame.download(outFrame);
	
	//Free dilatedFrame pointer device memory
	hipFree(d_erodedPtr);
	d_erodedFrame.release();
	
	return outFrame;
	//return *outFrame;
	//For the sake of debugging 
	return frame;
}

Mat averageFrame(Mat buffer[3]) {
	int threadCount = 1024;
	int blocks = (X * Y - 1) / threadCount + 1;
	if (blocks == 1)
	{
		threadCount = X * Y;
	}
	
	//copy buffer frames to device memory GpuMat
	//cv::cuda::GpuMat d_bufferFrames[3];
	//d_bufferFrames[0].upload(buffer[0]);
	//d_bufferFrames[1].upload(buffer[1]);
	//d_bufferFrames[2].upload(buffer[2]);
	
	
	//cv::cuda::PtrStepSz<float> *bufferPtr;
	//cv::cuda::PtrStepSz<float> d_arr[3];
	cv::cuda::PtrStepSz<uint8_t[3]> *bufferPtr;
	cv::cuda::PtrStepSz<uint8_t[3]> d_arr[3];
	cv::cuda::GpuMat d_bufferFrames[3];
	for (int i = 0; i < 3; i++) {
		d_bufferFrames[i].upload(buffer[i]);
		d_arr[i] = d_bufferFrames[i];
	}
	hipMalloc((void**)&bufferPtr, sizeof(cv::cuda::PtrStepSz<uint8_t[3]>) * 3);
	hipMemcpy(bufferPtr, d_arr, sizeof(cv::cuda::PtrStepSz<uint8_t[3]>) * 3, hipMemcpyHostToDevice);
	
	//hipMalloc((void**)&bufferPtr, sizeof(cv::cuda::PtrStepSz<float>)*3);
	//hipMemcpy(bufferPtr, d_arr, sizeof(cv::cuda::PtrStepSz<float>) * 3, hipMemcpyHostToDevice);

	uint8_t *d_bufferPtr;
	uint8_t *d_outPtr;
	cv::cuda::GpuMat d_outFrame;
	d_bufferFrames[0].copyTo(d_outFrame);

	hipMalloc((void **)&d_outPtr, d_outFrame.rows*d_outFrame.step);
	hipMemcpyAsync(d_outPtr, d_outFrame.ptr<uint8_t>(), d_outFrame.rows*d_outFrame.step, hipMemcpyDeviceToDevice);

	//convert the frame to be completely black to avoid weird artifacts
	blackKernel<<<blocks, threadCount>>>(d_outFrame);

	//allocate new device memory
	hipMalloc((void**)&d_bufferPtr, 3*d_bufferFrames[0].rows*d_bufferFrames[0].step);
	hipMemcpyAsync(d_bufferPtr, d_bufferFrames[0].ptr<uint8_t>(), 3 * d_bufferFrames[0].rows*d_bufferFrames[0].step, hipMemcpyDeviceToDevice);

	averageKernel<<<blocks, threadCount>>>(d_outFrame, bufferPtr);//d_bufferFrames);

	//free buffer pointer from device memory
	hipFree(d_bufferPtr);
	d_bufferFrames[0].release();
	d_bufferFrames[1].release();
	d_bufferFrames[2].release();

	Mat outFrame;
	//d_outFrame.download(outFrame);
	//free out pointer from device memory
	hipFree(d_outPtr);
	d_outFrame.release();


	//return outFrame;
	//for the sake of debugging
	return buffer[0];
}