#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

//#include <opencv2/cudev/ptr2d/gpumat.hpp>
//Canny and Gaussian
#include <opencv2/cudaimgproc.hpp>
#include <opencv2/videoio.hpp>
#include <opencv2/highgui.hpp>
#include <iostream>
using namespace std;
#include <opencv2/core.hpp>
using namespace cv;
#include <opencv2/cudaarithm.hpp>
using namespace cv::cuda;

__global__ void trackKernel(int *c, const int *a, const int *b)
{
	//detect end
	//determine orientation
	//determine distance
	//have internal representation of it's position in 3D
	//draw particles in 3D space
	//move particles with physics based on the match's movement
}

int main()
{
	//test code from Get started with OpenCV CUDA cpp
	printShortCudaDeviceInfo(getDevice());
	int cuda_devices_number = getCudaEnabledDeviceCount();
	cout << "CUDA Device(s) Number: " << cuda_devices_number << endl;
	DeviceInfo _deviceInfo;
	bool _isd_evice_compatible = _deviceInfo.isCompatible();
	cout << "CUDA Device(s) Compatible: " << _isd_evice_compatible << endl;

	//Mat frame;
	//VideoCapture cam;// = VideoCapture(0, CAP_DSHOW);
	//cam.open(0);// CAP_DSHOW);
	//if (!cam.isOpened()) {
	//	cerr << "ERROR Unable to open camera\n";
	//	return -1;
	//}
	//cam.read(frame);
	//if (frame.empty()) {
	//	cerr << "ERROR Blank frame\n";
	//	return -1;
	//}
	//destroyAllWindows();


	//working code that opens and displays the webcam
	/*
	VideoCapture cap(0);
	//cap.open(device);
	//cap.set(CV_CAP_PROP_FOURCC, CV_FOURCC(�eM�f, �eJ�f, �eP�f, �eG�f));
	cap.set(CV_CAP_PROP_FRAME_WIDTH, 1280);
	cap.set(CV_CAP_PROP_FRAME_HEIGHT, 720);
	int frameN = 0;
	while (1)
	{
		Mat frame;
		cap >> frame;
		if (!frame.data) break;
		//if (waitKey(30) >= 0) break;

		imshow("Camera", frame);
		if (frameN%30 == 0){
			printf("%d\n", frameN);
		}
		frameN++;
		waitKey(1);
	}*/
	
	VideoCapture cap(0); // open the default camera
	if (!cap.isOpened())  // check if we succeeded
		return -1;

	Mat edges;
	//namedWindow("edges", 1);
	for (;;)
	{
		Mat frame;
		cap >> frame; // get a new frame from camera
		//cv::cuda::cvtColor(frame, edges, CV_BGR2GRAY);
		//GaussianBlur(edges, edges, Size(7, 7), 1.5, 1.5);
		//Canny(edges, edges, 0, 30, 3);
		//imshow("edges", edges);
		//if (waitKey(30) >= 0) break;
		//waitKey(1);
	}
	// the camera will be deinitialized automatically in VideoCapture destructor
	//return 0;
	

	//Old basic CUDA code sample
	//--- INITIALIZE VIDEOCAPTURE
	//VideoCapture cap;
	// open the default camera using default API
	// cap.open(0);
	// OR advance usage: select any API backend
	//int deviceID = 0;             // 0 = open default camera
	//int apiID = cv::CAP_ANY;      // 0 = autodetect default API
	// open selected camera using selected API
	//cap.open(deviceID, apiID);
	// check if we succeeded
	/*if (!cap.isOpened()) {
		cerr << "ERROR! Unable to open camera\n";
		return -1;
	}
	
	//--- GRAB AND WRITE LOOP
	cout << "Start grabbing" << endl
		<< "Press any key to terminate" << endl;
	for (;;)
	{
		// wait for a new frame from camera and store it into 'frame'
		cap.read(frame);
		// check if we succeeded
		if (frame.empty()) {
			cerr << "ERROR! blank frame grabbed\n";
			break;
		}
		// show live and wait for a key with timeout long enough to show images
		imshow("Live", frame);
		if (waitKey(5) >= 0)
			break;
	}*/

	return 0;
}