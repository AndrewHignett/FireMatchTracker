#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include "matchTracker.h"
#include <opencv2/cudaarithm.hpp>
using namespace cv::cuda;

#define X 1280
#define Y 720

__global__ void trackKernel(cv::cuda::GpuMat out, cv::cuda::GpuMat frame)
{
	//detect end
	//determine orientation
	//determine distance
	//have internal representation of it's position in 3D
	//draw particles in 3D space
	//move particles with physics based on the match's movement
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadId < X * Y)
	{
		int row = threadId / X;
		int column = threadId % X;
		//BGR pixel values
		uint8_t pixelB = frame.data[(row*frame.step) + column * 3];
		uint8_t pixelG = frame.data[(row*frame.step) + column * 3 + 1];
		uint8_t pixelR = frame.data[(row*frame.step) + column * 3 + 2];
		//out.data[(row*out.step) + column * 3] = pixelB;
		//out.data[(row*out.step) + column * 3 + 1] = pixelG;
		//out.data[(row*out.step) + column * 3 + 2] = pixelR;
		if ((pixelR > 128) && (pixelB < 50) && (pixelG < 50))
		{
			out.data[(row*out.step) + column * 3] = pixelB;
			out.data[(row*out.step) + column * 3 + 1] = pixelG;
			out.data[(row*out.step) + column * 3 + 2] = pixelR;
		}
		else
		{
			out.data[(row*out.step) + column * 3] = 0;
			out.data[(row*out.step) + column * 3 + 1] = 0;
			out.data[(row*out.step) + column * 3 + 2] = 0;
		}
	}
}

Mat track(Mat frame) {
	//Mat *newFrame = (Mat*)malloc(X * Y * sizeof(Mat));
	//Mat *newFrame = (Mat*)malloc(sizeof(frame));
	//Mat *outFrame = (Mat*)malloc(sizeof(frame));
	//*newFrame = frame.clone();
	int threadCount = 1024;
	int blocks = (X * Y - 1) / threadCount + 1;
	if (blocks == 1)
	{
		threadCount = X * Y;
	}
	//Set up device variables
	//Mat *d_newFrame;
	//Mat *d_outFrame;
	uint8_t *d_imgPtr;
	uint8_t *d_outPtr;
	cv::cuda::GpuMat d_newFrame;
	cv::cuda::GpuMat d_outFrame;
	d_newFrame.upload(frame);
	d_outFrame.upload(frame);
	//Allocate device memory
	hipMalloc((void **)&d_imgPtr, d_newFrame.rows*d_newFrame.step);
	hipMalloc((void **)&d_outPtr, d_outFrame.rows*d_outFrame.step);
	hipMemcpyAsync(d_imgPtr, d_newFrame.ptr<uint8_t>(), d_newFrame.rows*d_newFrame.step, hipMemcpyDeviceToDevice);
	hipMemcpyAsync(d_outPtr, d_outFrame.ptr<uint8_t>(), d_outFrame.rows*d_outFrame.step, hipMemcpyDeviceToDevice);
	//hipMalloc((void**)&d_newFrame, sizeof(frame));
	//hipMalloc((void**)&d_outFrame, sizeof(frame));
	//transfer memory from host to device memory
	//hipMemcpy(d_newFrame, newFrame, sizeof(frame), hipMemcpyHostToDevice);
	trackKernel<<<blocks, threadCount>>>(d_outFrame, d_newFrame);
	//Free newFrame device and host memory
	//hipFree(d_newFrame);
	//free(newFrame);
	//hipMemcpy(outFrame, d_outFrame, sizeof(frame), hipMemcpyDeviceToHost);
	//Free outFrame device memory
	hipFree(d_imgPtr);
	hipFree(d_outPtr);
	
	Mat outFrame;
	d_outFrame.download(outFrame);
	return outFrame;
	//return *outFrame;
	//For the sake of debugging 
	return frame;
}